#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>

#define DEFAULT_MATRIX_SIZE 256
#define DEFAULT_BLOCK_SIZE 256

void printUsage(char* argv[]) {
    printf("Usage: %s <matrix_size> <:OPTIONAL: threads_per_block>", argv[0]);
}

__global__ void initializePRNGKernel(hiprandState *state, int seed, int maxIndex){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < maxIndex) {
        hiprand_init((seed << 20) + idx, 0, 0, &state[idx]);
    }
}

__global__ void getRandomMatrixKernel(float* resultMatrix, hiprandState *state, int maxIndex) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < maxIndex) {
        resultMatrix[idx] = 255.0 * hiprand_uniform(&state[idx]);
    }
}

float makeRandomMatrix(float* inputMatrix, int matrixSize, int blockSize) {
    int totalElements = matrixSize * matrixSize;
    int seed = time(0);
    int numBlocks = (totalElements + blockSize - 1) / blockSize;

    // allocate resources
    hiprandState *rngState;
    hipMalloc(&rngState, totalElements * sizeof(*rngState));
    float *deviceMatrix;
    hipMalloc(&deviceMatrix, totalElements * sizeof(*deviceMatrix));

    // setup timing
    float kernelTime;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // setup the rng
    initializePRNGKernel<<<numBlocks, blockSize>>>(rngState, seed, totalElements);
    hipDeviceSynchronize();

    // generate random matrix and copy it back
    getRandomMatrixKernel<<<numBlocks, blockSize>>>(deviceMatrix, rngState, totalElements);
    hipDeviceSynchronize();
    hipMemcpy(inputMatrix, deviceMatrix, totalElements * sizeof(*inputMatrix), hipMemcpyDeviceToHost);

    // stop the clock
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&kernelTime, start, stop);

    // free resources
    hipFree(rngState);
    hipFree(deviceMatrix);

    // return time
    return kernelTime;
}

float getSingularValues(float* inputMatrix, int matrixSize, int blockSize) {
    // setup solver handles
    hipsolverHandle_t cusolverH = NULL;
    hipblasHandle_t cublasH = NULL;

    hipsolverDnCreate(&cusolverH);
    hipblasCreate(&cublasH);

    // constant dimensions for column major matrix
    const int m = matrixSize;
    const int n = matrixSize;
    const int lda = m;

    // result array
    float *S = NULL; // [n] singular values
    hipHostMalloc(&S, n * (sizeof(*S)));

    // setup device arrays
    float *d_A = NULL;
    float *d_S = NULL;
    float *d_U = NULL;
    float *d_VT = NULL;
    int *devInfo = NULL;
    float *d_work = NULL;
    float *d_rwork = NULL;

    hipMalloc(&d_A, sizeof(float) * lda * n);
    hipMalloc(&d_S, sizeof(float) * n);
    hipMalloc(&d_U, sizeof(float) * lda * m);
    hipMalloc(&d_VT, sizeof(float) * lda * n);
    hipMalloc(&devInfo, sizeof(int));

    // setup timing
    float kernelTime;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // copy input array to device
    hipMemcpy(d_A, inputMatrix, sizeof(float) * lda * n, hipMemcpyHostToDevice);

    // query working space of SVD solver
    int lwork = 0;
    hipsolverDnSgesvd_bufferSize(cusolverH, m, n, &lwork);
    hipMalloc(&d_work, sizeof(float) * lwork);

    // compute SVD
    signed char jobu = 'A'; // all m columns of U
    signed char jobvt = 'A'; // all n columns of VT
    hipsolverDnSgesvd(
            cusolverH,
            jobu,
            jobvt,
            m,
            n,
            d_A,
            lda,
            d_S,
            d_U,
            lda,  // ldu
            d_VT,
            lda, // ldvt,
            d_work,
            lwork,
            d_rwork,
            devInfo);

    hipDeviceSynchronize();

    // copy singular value results back
    hipMemcpy(S, d_S, sizeof(float) * n, hipMemcpyDeviceToHost);

    // stop the clock
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&kernelTime, start, stop);

    // free resources
    hipblasDestroy(cublasH);
    hipsolverDnDestroy(cusolverH);

    hipFree(devInfo);
    hipFree(d_rwork);
    hipFree(d_work);

    hipFree(d_A);
    hipFree(d_S);
    hipFree(d_U);
    hipFree(d_VT);

    hipHostFree(S);

    return kernelTime;
}

int main(int argc, char* argv[]) {
    // parse args
    if(argc > 3) {
        printUsage(argv);
	return -1;
    }

    int matrixSize = DEFAULT_MATRIX_SIZE;
    if(argc > 1) {
        matrixSize = atoi(argv[1]);
    }

    int blockSize = DEFAULT_BLOCK_SIZE;
    if(argc > 2) {
        blockSize = atoi(argv[2]);
    }

    // allocate space for the matrix
    float* randomMatrix;
    hipHostMalloc(&randomMatrix, matrixSize * matrixSize * sizeof(*randomMatrix));

    // use hiprand to generate random matrix
    float randomTime = makeRandomMatrix(randomMatrix, matrixSize, blockSize);

    // use cusolver to find singular values of the random matrix
    float svdTime = getSingularValues(randomMatrix, matrixSize, blockSize);

    // print results
    printf("Matrix Dimension: %d -- Threads Per Block: %d\n", matrixSize, blockSize);
    printf("PRNG operation time... : %f (ms)\n", randomTime);
    printf("Singular value operation time... : %f (ms)\n", svdTime);

    // free remaining resources
    hipFree(randomMatrix);

    return 0;
}
